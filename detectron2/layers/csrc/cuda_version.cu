// Copyright (c) Facebook, Inc. and its affiliates. All Rights Reserved.


#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

namespace detectron2 {
int get_cudart_version() {
#if __HIP_PLATFORM_NVCC__ 
  return CUDART_VERSION;
#else
  return 0;
#endif
}
} // namespace detectron2
